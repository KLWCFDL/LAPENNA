
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 10 CUDA threads running on the GPU.
 */

/*
 The qualifier "__global__" tells the compiler that the function will be called
 from the CPU and executed on the GPU
*/
__global__ void helloFromGPU()
{ /**/
  printf("Hello World from GPU!\n");
}

int main(int argc, char **argv)
{
  printf("Hello World from CPU!\n");

  helloFromGPU<<<1, 10>>>();

  hipDeviceReset();
  return 0;
}
