
#include "image_io.h"
#include <cassert>
#include <cstdlib>
#include <hipDNN.h>
#include <iostream>

/*The implementation of a basic convolution operation with cuDNN
 *reference:http://www.goldsborough.me/cuda/ml/cudnn/c++/2017/10/01/14-37-23-convolutions_with_cudnn/
 *https://gist.github.com/goldsborough/865e6717e64fbae75cdaf6c9914a130d
 */

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

int main(int argc, const char *argv[])
{
  if (argc < 2) {
    std::cerr << "usage: conv <image> [gpu=0] [sigmoid=0]" << std::endl;
    std::exit(EXIT_FAILURE);
  }

  int gpu_id = (argc > 2) ? std::atoi(argv[2]) : 0;
  std::cerr << "GPU: " << gpu_id << std::endl;

  bool with_sigmoid = (argc > 3) ? std::atoi(argv[3]) : 0;
  std::cerr << "With sigmoid: " << std::boolalpha << with_sigmoid << std::endl;

  int width, height, c;
  float *image = read_image(argv[1], &c, &height, &width);

  hipSetDevice(gpu_id);

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/3,
                                        /*channels=*/3,
                                        /*image_height=*/height,
                                        /*image_width=*/width));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*out_channels=*/3,
                                        /*in_channels=*/3,
                                        /*kernel_height=*/3,
                                        /*kernel_width=*/3));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/1,
                                             /*pad_width=*/1,
                                             /*vertical_stride=*/1,
                                             /*horizontal_stride=*/1,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION,
                                             /*computeType=*/HIPDNN_DATA_FLOAT));

  int batch_size{0}, out_channels{0}, out_height{0}, out_width{0};
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(
      convolution_descriptor, input_descriptor, kernel_descriptor, &batch_size,
      &out_channels, &out_height, &out_width));

  std::cerr << "Output Image: " << batch_size << " x " << out_height << " x "
            << out_width << " x " << out_channels << std::endl;

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/3,
                                        /*channels=*/3,
                                        /*image_height=*/out_height,
                                        /*image_width=*/out_width));

  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(
      cudnn, input_descriptor, kernel_descriptor, convolution_descriptor,
      output_descriptor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
      /*memoryLimitInBytes=*/0, &convolution_algorithm));

  size_t workspace_bytes{0};
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
      cudnn, input_descriptor, kernel_descriptor, convolution_descriptor,
      output_descriptor, convolution_algorithm, &workspace_bytes));
  std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
            << std::endl;
  assert(workspace_bytes > 0);

  void *d_workspace{nullptr};
  hipMalloc(&d_workspace, workspace_bytes);

  int image_bytes =
      batch_size * out_channels * out_height * out_width * sizeof(float);

  printf("batch size: %i\n", batch_size);

  float *d_input{nullptr};
  hipMalloc(&d_input, image_bytes);
  hipMemcpy(d_input, image, image_bytes, hipMemcpyHostToDevice);

  float *d_output{nullptr};
  hipMalloc(&d_output, image_bytes);
  hipMemset(d_output, 0, image_bytes);

  // clang-format off
  float edges = -1;
  float corners = -1;
  const float kernel_template[3][3] = {
    {corners, edges, corners},
    {edges, 8, edges},
    {corners, edges, corners}
  };
  // clang-format on

  float h_kernel[3][3][3][3];
  for (int kernel = 0; kernel < 3; ++kernel) {
    for (int channel = 0; channel < 3; ++channel) {
      for (int row = 0; row < 3; ++row) {
        for (int column = 0; column < 3; ++column) {
          h_kernel[kernel][channel][row][column] = kernel_template[row][column];
        }
      }
    }
  }

  float *d_kernel{nullptr};
  hipMalloc(&d_kernel, sizeof(h_kernel));
  hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

  const float alpha = 1.0 / 3.0, beta = 0.0f;  // why do we need to divide by 3?

  checkCUDNN(hipdnnConvolutionForward(
      cudnn, &alpha, input_descriptor, d_input, kernel_descriptor, d_kernel,
      convolution_descriptor, convolution_algorithm, d_workspace,
      workspace_bytes, &beta, output_descriptor, d_output));

  if (with_sigmoid) {
    hipdnnActivationDescriptor_t activation_descriptor;
    checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
    checkCUDNN(hipdnnSetActivationDescriptor(
        activation_descriptor, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN,
        /*relu_coef=*/0));
    checkCUDNN(hipdnnActivationForward(cudnn, activation_descriptor, &alpha,
                                      output_descriptor, d_output, &beta,
                                      output_descriptor, d_output));
    hipdnnDestroyActivationDescriptor(activation_descriptor);
  }

  float *h_output = new float[image_bytes];
  hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);

  write_image(argv[1], h_output, out_channels, out_height, out_width);

  delete[] h_output;
  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);

  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

  hipdnnDestroy(cudnn);
}
