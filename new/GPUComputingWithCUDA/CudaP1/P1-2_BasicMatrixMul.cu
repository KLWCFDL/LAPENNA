
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

/*
 * This example helps to visualize the relationship between thread/block IDs and
 * offsets into data. For each CUDA thread, this example displays the
 * intra-block thread ID, the inter-block block ID, the global coordinate of a
 * thread, the calculated offset into input data, and the input data at that
 * offset.
 */

__global__ void matrixMult(int *a, int *b, int *c, int width)
{
  int k, sum = 0;
  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;
  if (col < width && row < width) {
    for (k = 0; k < width; k++)
      sum += a[row * width + k] * b[k * width + col];
    c[row * width + col] = sum;
  }
}

int main()
{
  int a[N][N], b[N][N], c[N][N];
  int *dev_a, *dev_b, *dev_c;
  int size = N * N * sizeof(int);

  hipMalloc((void **)&dev_a, size);
  hipMalloc((void **)&dev_b, size);
  hipMalloc((void **)&dev_c, size);

  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

  dim3 dimGrid(1, 1);
  dim3 dimBlock(N, N);

  matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);
  hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}
