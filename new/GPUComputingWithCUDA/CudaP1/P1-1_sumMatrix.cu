
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. A 2D thread block and 2D grid are used. sumArraysOnHost sequentially
 * iterates through vector elements on the host.
 */

/*Initializes a random array of floats*/
void initialData(float *ip, const int size)
{
  int i;

  for (i = 0; i < size; i++) {
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }

  return;
}

/*Sums two matricies on CPU to check the GPU results*/
void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
  float *ia = A;
  float *ib = B;
  float *ic = C;

  for (int iy = 0; iy < ny; iy++) {
    for (int ix = 0; ix < nx; ix++) {
      ic[ix] = ia[ix] + ib[ix];
    }

    ia += nx;
    ib += nx;
    ic += nx;
  }

  return;
}

/*determines if two floats are within epsilon of each other*/
void checkResult(float *hostRef, float *gpuRef, const int N)
{
  double epsilon = 1.0E-8;
  bool match = 1;

  for (int i = 0; i < N; i++) {
    if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
      match = 0;
      printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
      break;
    }
  }

  if (match)
    printf("Arrays match.\n\n");
  else
    printf("Arrays do not match.\n\n");
}

// sums a portion of the matrix on the GPU given it's thread id
__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx,
                                 int ny)
{
  unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int idx = iy * nx + ix;

  if (ix < nx && iy < ny)
    MatC[idx] = MatA[idx] + MatB[idx];
}

int main(int argc, char **argv)
{
  printf("%s Starting...\n", argv[0]);

  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  hipSetDevice(dev);

  // set up data size of matrix
  int nx = 1 << 14;
  int ny = 1 << 14;

  int nxy = nx * ny;
  int nBytes = nxy * sizeof(float);
  printf("Matrix size: nx %d ny %d\n", nx, ny);

  // malloc host memory
  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef = (float *)malloc(nBytes);

  // initialize data at host side
  initialData(h_A, nxy);
  initialData(h_B, nxy);

  memset(hostRef, 0, nBytes);
  memset(gpuRef, 0, nBytes);

  // add matrix at host side for result checks
  sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);

  // malloc device global memory
  float *d_MatA, *d_MatB, *d_MatC;
  hipMalloc((void **)&d_MatA, nBytes);
  hipMalloc((void **)&d_MatB, nBytes);
  hipMalloc((void **)&d_MatC, nBytes);

  // transfer data from host to device
  hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

  // invoke kernel at host side
  int dimx = 32;
  int dimy = 32;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
  hipDeviceSynchronize();
  // check kernel error
  hipGetLastError();

  // copy kernel result back to host side
  hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);

  // check device results
  checkResult(hostRef, gpuRef, nxy);

  // free device global memory
  hipFree(d_MatA);
  hipFree(d_MatB);
  hipFree(d_MatC);

  // free host memory
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  // reset device
  hipDeviceReset();

  return (0);
}
