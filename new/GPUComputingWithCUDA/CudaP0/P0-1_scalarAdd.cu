
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This program performs scalar addition "a+b=c" on the GPU.
 */

__global__ void add(int *a, int *b, int *c)
{ /**/
  *c = *a + *b;
}

int main(void)
{
  int a, b, c;           // host copies of a, b, c
  int *d_a, *d_b, *d_c;  // memory locations of device copies of a, b, c
  int size = sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Setup input values
  a = 2;
  b = 7;
  // Copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU
  add<<<1, 1>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  printf("%i + %i = %i\n", a, b, c);

  return 0;
}
