
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Various memory access pattern optimizations applied to a matrix transpose
 * kernel.
 */

#define BDIMX 16
#define BDIMY 16

void initialData(float *in, const int size)
{
  for (int i = 0; i < size; i++) {
    in[i] = (float)(rand() & 0xFF) / 10.0f;  // 100.0f;
  }

  return;
}

void printData(float *in, const int size)
{
  for (int i = 0; i < size; i++) {
    printf("%dth element: %f\n", i, in[i]);
  }

  return;
}

void checkResult(float *hostRef, float *gpuRef, const int size, int showme)
{
  double epsilon = 1.0E-8;
  bool match = 1;

  for (int i = 0; i < size; i++) {
    if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
      match = 0;
      printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
             gpuRef[i]);
      break;
    }

    if (showme && i > size / 2 && i < size / 2 + 5) {
      // printf("%dth element: host %f gpu %f\n",i,hostRef[i],gpuRef[i]);
    }
  }

  if (!match)
    printf("Arrays do not match.\n\n");
}

void transposeHost(float *out, float *in, const int nx, const int ny)
{
  for (int iy = 0; iy < ny; ++iy) {
    for (int ix = 0; ix < nx; ++ix) {
      out[ix * ny + iy] = in[iy * nx + ix];
    }
  }
}

__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

  if (ix < nx && iy < ny) {
    out[iy * nx + ix] = in[iy * nx + ix];
  }
}

// case 0 copy kernel: access data in rows
__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

  if (ix < nx && iy < ny) {
    out[iy * nx + ix] = in[iy * nx + ix];
  }
}

// case 1 copy kernel: access data in columns
__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

  if (ix < nx && iy < ny) {
    out[ix * ny + iy] = in[ix * ny + iy];
  }
}

// case 2 transpose kernel: read in rows and write in columns
__global__ void transposeNaiveRow(float *out, float *in, const int nx,
                                  const int ny)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

  if (ix < nx && iy < ny) {
    out[ix * ny + iy] = in[iy * nx + ix];
  }
}

// case 3 transpose kernel: read in columns and write in rows
__global__ void transposeNaiveCol(float *out, float *in, const int nx,
                                  const int ny)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

  if (ix < nx && iy < ny) {
    out[iy * nx + ix] = in[ix * ny + iy];
  }
}

// case 4 transpose kernel: read in rows and write in columns + unroll 4 blocks
__global__ void transposeUnroll4Row(float *out, float *in, const int nx,
                                    const int ny)
{
  unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
  unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

  unsigned int ti = iy * nx + ix;  // access in rows
  unsigned int to = ix * ny + iy;  // access in columns

  if (ix + 3 * blockDim.x < nx && iy < ny) {
    out[to] = in[ti];
    out[to + ny * blockDim.x] = in[ti + blockDim.x];
    out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
    out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
  }
}

// case 5 transpose kernel: read in columns and write in rows + unroll 4 blocks
__global__ void transposeUnroll4Col(float *out, float *in, const int nx,
                                    const int ny)
{
  unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
  unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

  unsigned int ti = iy * nx + ix;  // access in rows
  unsigned int to = ix * ny + iy;  // access in columns

  if (ix + 3 * blockDim.x < nx && iy < ny) {
    out[ti] = in[to];
    out[ti + blockDim.x] = in[to + blockDim.x * ny];
    out[ti + 2 * blockDim.x] = in[to + 2 * blockDim.x * ny];
    out[ti + 3 * blockDim.x] = in[to + 3 * blockDim.x * ny];
  }
}

/*
 * case 6 :  transpose kernel: read in rows and write in colunms + diagonal
 * coordinate transform
 */
__global__ void transposeDiagonalRow(float *out, float *in, const int nx,
                                     const int ny)
{
  unsigned int blk_y = blockIdx.x;
  unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

  unsigned int ix = blockDim.x * blk_x + threadIdx.x;
  unsigned int iy = blockDim.y * blk_y + threadIdx.y;

  if (ix < nx && iy < ny) {
    out[ix * ny + iy] = in[iy * nx + ix];
  }
}

/*
 * case 7 :  transpose kernel: read in columns and write in row + diagonal
 * coordinate transform.
 */
__global__ void transposeDiagonalCol(float *out, float *in, const int nx,
                                     const int ny)
{
  unsigned int blk_y = blockIdx.x;
  unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

  unsigned int ix = blockDim.x * blk_x + threadIdx.x;
  unsigned int iy = blockDim.y * blk_y + threadIdx.y;

  if (ix < nx && iy < ny) {
    out[iy * nx + ix] = in[ix * ny + iy];
  }
}

// main functions
int main(int argc, char **argv)
{
  if (argc != 2) {
    fprintf(stderr, "Please Specify a kernel to run.\n"
                    "(ex. ./P2-0_MatixTranspose 2)\n"
                    "0 = copyRow\n"
                    "1 = copyCol\n"
                    "2 = transposeNaiveRow\n"
                    "3 = transposeNaiveCol\n"
                    "4 = transposeUnroll4Row\n"
                    "5 = transposeUnroll4Col\n"
                    "6 = transposeDiagonalRow\n"
                    "7 = transposeDiagonalCol\n");
    return 1;
  }
  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("%s starting transpose at ", argv[0]);
  printf("device %d: %s ", dev, deviceProp.name);
  hipSetDevice(dev);

  // set up array size 2048
  int nx = 1 << 11;
  int ny = 1 << 11;

  // select a kernel and block size
  int iKernel = atoi(argv[1]);
  int blockx = 16;
  int blocky = 16;

  printf(" with matrix nx %d ny %d with kernel %d\n", nx, ny, iKernel);
  size_t nBytes = nx * ny * sizeof(float);

  // execution configuration
  dim3 block(blockx, blocky);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  // allocate host memory
  float *h_A = (float *)malloc(nBytes);
  float *hostRef = (float *)malloc(nBytes);
  float *gpuRef = (float *)malloc(nBytes);

  // initialize host array
  initialData(h_A, nx * ny);

  // transpose at host side
  transposeHost(hostRef, h_A, nx, ny);

  // allocate device memory
  float *d_A, *d_C;
  hipMalloc((float **)&d_A, nBytes);
  hipMalloc((float **)&d_C, nBytes);

  // copy data from host to device
  hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);

  // warmup to avoide startup overhead

  warmup<<<grid, block>>>(d_C, d_A, nx, ny);
  hipDeviceSynchronize();

  hipGetLastError();

  // kernel pointer and descriptor
  void (*kernel)(float *, float *, int, int);

  // set up kernel
  switch (iKernel) {
    case 0:
      kernel = &copyRow;
      break;

    case 1:
      kernel = &copyCol;
      break;

    case 2:
      kernel = &transposeNaiveRow;
      break;

    case 3:
      kernel = &transposeNaiveCol;
      break;

    case 4:
      kernel = &transposeUnroll4Row;
      grid.x = (nx + block.x * 4 - 1) / (block.x * 4);
      break;

    case 5:
      kernel = &transposeUnroll4Col;
      grid.x = (nx + block.x * 4 - 1) / (block.x * 4);
      break;

    case 6:
      kernel = &transposeDiagonalRow;
      break;

    case 7:
      kernel = &transposeDiagonalCol;
      break;
  }

  // run kernel

  kernel<<<grid, block>>>(d_C, d_A, nx, ny);
  hipDeviceSynchronize();

  hipGetLastError();

  // check kernel results
  if (iKernel > 1) {
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    checkResult(hostRef, gpuRef, nx * ny, 1);
  }

  // free host and device memory
  hipFree(d_A);
  hipFree(d_C);
  free(h_A);
  free(hostRef);
  free(gpuRef);

  // reset device
  hipDeviceReset();
  return EXIT_SUCCESS;
}
